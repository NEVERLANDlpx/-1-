﻿#include <iostream>
#include <fstream>
#include <sstream>
#include <map>
#include <vector>
#include <hip/hip_runtime.h>

using namespace std;

#define NUM_THREADS 8
const int maxsize = 3000;
const int maxrow = 40000;
const int numBasis = 40000;
int num;

vector<int> tmpAns;
long long head, tail, freq;
map<int, int*> ans;

fstream RowFile("被消元行.txt", ios::in | ios::out);
fstream BasisFile("消元子.txt", ios::in | ios::out);

ofstream out_mpi("消元结果(MPI).txt");

int gRows[maxrow][maxsize];
int gBasis[numBasis][maxsize];
int answers[maxrow][maxsize];
map<int, int> firstToRow;

int ifBasis[numBasis] = { 0 };
int ifDone[maxrow] = { 0 };

void reset() {
    memset(gRows, 0, sizeof(gRows));
    memset(gBasis, 0, sizeof(gBasis));
    memset(ifBasis, 0, sizeof(ifBasis));
    RowFile.close();
    BasisFile.close();
    RowFile.open("被消元行.txt", ios::in | ios::out);
    BasisFile.open("消元子.txt", ios::in | ios::out);
    ans.clear();
}

int readBasis() {
    for (int i = 0; i < numBasis; i++) {
        if (BasisFile.eof()) {
            cout << "读取消元子" << i - 1 << "行" << endl;
            return i - 1;
        }
        string tmp;
        bool flag = false;
        int row = 0;
        getline(BasisFile, tmp);
        stringstream s(tmp);
        int pos;
        while (s >> pos) {
            if (!flag) {
                row = pos;
                flag = true;
                ifBasis[row] = 1;
            }
            int index = pos / 32;
            int offset = pos % 32;
            gBasis[row][index] = gBasis[row][index] | (1 << offset);
        }
        flag = false;
        row = 0;
    }
}

int readRowsFrom(int pos) {
    if (RowFile.is_open())
        RowFile.close();
    RowFile.open("被消元行.txt", ios::in | ios::out);
    memset(gRows, 0, sizeof(gRows));
    string line;
    for (int i = 0; i < pos; i++) {
        getline(RowFile, line);
    }
    for (int i = pos; i < pos + maxrow; i++) {
        int tmp;
        getline(RowFile, line);
        if (line.empty()) {
            cout << "读取被消元行 " << i << " 行" << endl;
            return i;
        }
        bool flag = false;
        stringstream s(line);
        while (s >> tmp) {
            int index = tmp / 32;
            int offset = tmp % 32;
            gRows[i - pos][index] = gRows[i - pos][index] | (1 << offset);
            flag = true;
        }
    }
    cout << "read max rows" << endl;
    return -1;
}

int findfirst(int row) {
    int first;
    for (int i = maxsize - 1; i >= 0; i--) {
        if (gRows[row][i] == 0)
            continue;
        else {
            int pos = i * 32;
            int offset = 0;
            for (int k = 31; k >= 0; k--) {
                if (gRows[row][i] & (1 << k)) {
                    offset = k;
                    break;
                }
            }
            first = pos + offset;
            return first;
        }
    }
    return -1;
}

int _findfirst(int row) {
    int first;
    for (int i = maxsize - 1; i >= 0; i--) {
        if (answers[row][i] == 0)
            continue;
        else {
            int pos = i * 32;
            int offset = 0;
            for (int k = 31; k >= 0; k--) {
                if (answers[row][i] & (1 << k)) {
                    offset = k;
                    break;
                }
            }
            first = pos + offset;
            return first;
        }
    }
    return -1;
}

void writeResult(ofstream &out) {
    for (auto it = ans.rbegin(); it != ans.rend(); it++) {
        int* result = it->second;
        int max = it->first / 32 + 1;
        for (int i = max; i >= 0; i--) {
            if (result[i] == 0)
                continue;
            int pos = i * 32;
            for (int k = 31; k >= 0; k--) {
                if (result[i] & (1 << k)) {
                    out << k + pos << " ";
                }
            }
        }
        out << endl;
    }
}

__global__ void grobner_reduction_kernel(int *d_rows, int *d_basis, int *d_ifBasis, int *d_answers, int maxsize, int numRows) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numRows) {
        int row_start = idx * maxsize;
        while (true) {
            int first = -1;
            for (int i = maxsize - 1; i >= 0; i--) {
                if (d_rows[row_start + i] != 0) {
                    int pos = i * 32;
                    for (int k = 31; k >= 0; k--) {
                        if (d_rows[row_start + i] & (1 << k)) {
                            first = pos + k;
                            break;
                        }
                    }
                    break;
                }
            }
            if (first == -1) break;

            if (d_ifBasis[first] == 1) {
                for (int j = 0; j < maxsize; j++) {
                    d_rows[row_start + j] ^= d_basis[first * maxsize + j];
                }
            } else {
                for (int j = 0; j < maxsize; j++) {
                    d_basis[first * maxsize + j] = d_rows[row_start + j];
                    d_answers[row_start + j] = d_rows[row_start + j];
                }
                d_ifBasis[first] = 1;
                break;
            }
        }
    }
}

void GE_GPU(int argc, char* argv[]) {
    int flag;
    double start_time = 0;
    double end_time = 0;

    int *d_rows, *d_basis, *d_ifBasis, *d_answers;
    hipMalloc(&d_rows, maxrow * maxsize * sizeof(int));
    hipMalloc(&d_basis, numBasis * maxsize * sizeof(int));
    hipMalloc(&d_ifBasis, numBasis * sizeof(int));
    hipMalloc(&d_answers, maxrow * maxsize * sizeof(int));

    flag = readRowsFrom(0);
    num = (flag == -1) ? maxrow : flag;

    hipMemcpy(d_rows, gRows, maxrow * maxsize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_basis, gBasis, numBasis * maxsize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ifBasis, ifBasis, numBasis * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_answers, answers, maxrow * maxsize * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (num + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    grobner_reduction_kernel<<<numBlocks, blockSize>>>(d_rows, d_basis, d_ifBasis, d_answers, maxsize, num);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "GPU优化版本耗时： " << milliseconds << "ms" << endl;

    hipMemcpy(gRows, d_rows, maxrow * maxsize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(gBasis, d_basis, numBasis * maxsize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(ifBasis, d_ifBasis, numBasis * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(answers, d_answers, maxrow * maxsize * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_rows);
    hipFree(d_basis);
    hipFree(d_ifBasis);
    hipFree(d_answers);

    for (int i = 0; i < num; i++) {
        if (findfirst(i) != -1) {
            ans.insert(pair<int, int*>(findfirst(i), gBasis[findfirst(i)]));
        }
    }
}

int main(int argc, char* argv[]) {
    ofstream out("消元结果.txt");

    readBasis();
    GE_GPU(argc, argv);
    writeResult(out);

    out.close();
    return 0;
}
